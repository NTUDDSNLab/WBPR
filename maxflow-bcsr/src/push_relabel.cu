#include "hip/hip_runtime.h"
#include"../include/parallel_graph.cuh"
#include "../include/utils.cuh"

#ifdef TIME_BREAKDOWN
void InitializeTimeBreakdown() {
    /* Initialize the time breakdown data */
    // Allocate device memory for scan time and backward time
    CHECK(hipMalloc((void**)&scanTime, totalWarps*sizeof(unsigned long long)));
    CHECK(hipMalloc((void**)&backwardTime, totalWarps*sizeof(unsigned long long)));
}

void FinializeTimeBreakdown() {
    /* Free the time breakdown data */
}


void report_breakdown_data(float totalExeTime) {
    /* Print the breakdown information to stdout */
    
    unsigned long long *scanTimeHost = (unsigned long long*)malloc(totalWarps*sizeof(unsigned long long));
    unsigned long long *backwardTimeHost = (unsigned long long*)malloc(totalWarps*sizeof(unsigned long long));

    unsigned long long *tempDeviceArray;
    CHECK(hipMalloc((void**)&tempDeviceArray, totalWarps*sizeof(unsigned long long)));

    CHECK(hipMemset(tempDeviceArray, 0, totalWarps * sizeof(unsigned long long)));

    if (scanTimeHost == NULL || backwardTimeHost == NULL) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return;
    }

    // Launch kernel using copyFromDeviceToHost() 
    copyScanToHost<<<numSM, numThreadsPerBlock>>>(tempDeviceArray, totalWarps);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipLaunchCooperativeKernel failed: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error, for example, by cleaning up resources and exiting
        exit(1);
    }
    
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(scanTimeHost, tempDeviceArray, totalWarps*sizeof(unsigned long long), hipMemcpyDeviceToHost));
    copyBackwardToHost<<<numSM, numThreadsPerBlock>>>(tempDeviceArray, totalWarps);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(backwardTimeHost, tempDeviceArray, totalWarps*sizeof(unsigned long long), hipMemcpyDeviceToHost));

    // CHECK(hipMemcpyFromSymbol(scanTimeHost, HIP_SYMBOL(scanTime), totalWarps*sizeof(unsigned long long)));
    // CHECK(hipMemcpyFromSymbol(backwardTimeHost, HIP_SYMBOL(backwardTime), totalWarps*sizeof(unsigned long long)));
    unsigned long long totalScanTime = 0;
    unsigned long long totalBackwardTime = 0;

    CHECK(hipDeviceSynchronize());


    // Calculate the maximum, std, mean, median, and min of scan time and backward time
    unsigned long long maxScanTime = 0;
    unsigned long long maxBackwardTime = 0;
    unsigned long long minScanTime = scanTimeHost[0];
    unsigned long long minBackwardTime = backwardTimeHost[0];
    float averageScanTime = 0;
    float averageBackwardTime = 0;

    for (int i = 0; i < totalWarps; i++) {
        if (scanTimeHost[i] > maxScanTime) {
            maxScanTime = scanTimeHost[i];
        }
        if (backwardTimeHost[i] > maxBackwardTime) {
            maxBackwardTime = backwardTimeHost[i];
        }
        if (scanTimeHost[i] < minScanTime) {
            minScanTime = scanTimeHost[i];
        }
        if (backwardTimeHost[i] < minBackwardTime) {
            minBackwardTime = backwardTimeHost[i];
        }
        totalScanTime += scanTimeHost[i];
        totalBackwardTime += backwardTimeHost[i];
    }

    averageScanTime = totalScanTime / totalWarps / 1000000;
    averageBackwardTime = totalBackwardTime / totalWarps / 100000;

    // Print averge time
    printf("Total execution time: %.6f ms\n", totalExeTime);
    printf("Average scan time: %.6f ms\n", averageScanTime);
    printf("Max scan time: %llu\n", maxScanTime/1000000);
    printf("Average backward time: %.6f ms\n", averageBackwardTime);
    printf("Max backward time: %llu\n", maxBackwardTime/100000);
    printf("Average other time: %.6f ms\n", totalExeTime - averageScanTime - averageBackwardTime);
   


    free(scanTimeHost);
    free(backwardTimeHost);
    CHECK(hipFree(tempDeviceArray));
}
#endif /* TIME_BREAKDOWN */




bool compare_excess_flow(int *new_excess_flow, int *old_excess_flow, int V)
{
    for(int i = 0; i < V; i++)
    {
        if (new_excess_flow[i] != old_excess_flow[i])
        {
            return false;
        }
    }
    return true;
}

void copy_excess_flow(int *new_excess_flow, int *old_excess_flow, int V)
{
    for(int i = 0; i < V; i++)
    {
        old_excess_flow[i] = new_excess_flow[i];
    }
}

void printExcessFlow(int V, int *excess_flow)
{
    printf("Excess flow values : \n");
    for(int i = 0; i < V; i++)
    {
        printf("%d ",excess_flow[i]);
    }
    printf("\n");
}


void push_relabel(int algo_type, int V, int E, int source, int sink, int *cpu_height, int *cpu_excess_flow, 
                int *cpu_offsets, int *cpu_destinations, int* cpu_capacities, int* cpu_fflows,
                int *Excess_total, 
                int *gpu_height, int *gpu_excess_flow, 
                int *gpu_offsets, int* gpu_destinations, int* gpu_capacities, int* gpu_fflows,
                int* gpu_avq, int* gpu_cycle)
{
    /* Instead of checking for overflowing vertices(as in the sequential push relabel),
     * sum of excess flow values of sink and source are compared against Excess_total 
     * If the sum is lesser than Excess_total, 
     * it means that there is atleast one more vertex with excess flow > 0, apart from source and sink
     */

    /* declaring the mark and scan boolean arrays used in the global_relabel routine outside the while loop 
     * This is not to lose the mark values if it goes out of scope and gets redeclared in the next iteration 
     */
    
    bool *mark,*scanned;
    mark = (bool*)malloc(V*sizeof(bool));
    scanned = (bool*)malloc(V*sizeof(bool));




    CudaTimer timer;
    float totalMilliseconds = 0.0f;
    printf("Inside push_relabel\n");


    // Configure the GPU
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    dim3 num_blocks(deviceProp.multiProcessorCount * numBlocksPerSM);
    dim3 block_size(numThreadsPerBlock);
    // dim3 num_blocks(1);
    // dim3 block_size(64);

    // Calculate the usage of shared memory
    size_t sharedMemSize = 3 * block_size.x * sizeof(int);

#ifdef WORKLOAD
    int max_iter = 1;
    int cur_iter = 0;
    // Caculate the total number of warps
    int num_warps = (block_size.x * num_blocks.x) / 32;
    
    // Allocate device buffer for warp execution time
    unsigned long long *gpu_warpExecutionTime;
    CHECK(hipMalloc((void**)&gpu_warpExecutionTime, num_warps*sizeof(unsigned long long)));

    // Allocate host buffer for warp execution time
    unsigned long long *cpuWarpExecution = (unsigned long long*)malloc(num_warps*sizeof(unsigned long long));
    unsigned long long *tempWarpExecution = (unsigned long long*)malloc(num_warps*sizeof(unsigned long long));
    for (int i = 0; i < num_warps; i++) {
        cpuWarpExecution[i] = 0;
    }
#endif // WORKLOAD

#ifdef TIME_BREAKDOWN
    InitializeTimeBreakdown();
#endif /* TIME_BREAKDOWN */

    // Print the configuration
    // Print GPU device name
    printf("GPU Device: %s\n", deviceProp.name);
    printf("Number of blocks: %d\n", num_blocks.x);
    printf("Number of threads per block: %d\n", block_size.x);
    printf("Total warps: %d\n", totalWarps);
    printf("Shared memory size: %lu\n", sharedMemSize);


    void* original_kernel_args[] = {&V, &source, &sink, &gpu_height, &gpu_excess_flow, 
                        &gpu_offsets, &gpu_destinations, &gpu_capacities, &gpu_fflows};


    void* kernel_args[] = {&V, &source, &sink, &gpu_height, &gpu_excess_flow, 
                        &gpu_offsets, &gpu_destinations, &gpu_capacities, &gpu_fflows, 
                        &gpu_avq, &gpu_cycle};


    // initialising mark values to false for all nodes
    for(int i = 0; i < V; i++)
    {
        mark[i] = false;
    }
    // for (int i = 0; i < 3; i++)
    while((cpu_excess_flow[source] + cpu_excess_flow[sink]) < *Excess_total)
    {
        printf("cpu_excess_flow[source]: %d, cpu_excess_flow[sink]: %d\n",cpu_excess_flow[source], cpu_excess_flow[sink]);

        //printf("gpu_excess_flow[source]: %d, gpu_excess_flow[sink]: %d\n",gpu_excess_flow[source], gpu_excess_flow[sink]);
        // copying height values to CUDA device global memory
        CHECK(hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpu_excess_flow, cpu_excess_flow, V*sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpu_fflows, cpu_fflows, E*sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemset(gpu_cycle, V, sizeof(int))); // Reset the gpu_cycle to V


        printf("Invoking kernel\n");

        hipError_t cudaStatus;
        timer.start();
        if (algo_type == 0) {
            // Thread-centric approach
            cudaStatus = hipLaunchCooperativeKernel((void*)push_relabel_kernel, num_blocks, block_size, original_kernel_args, sharedMemSize, 0);
        } else {
            // Vertex-centric approach
            cudaStatus = hipLaunchCooperativeKernel((void*)coop_push_relabel_kernel, num_blocks, block_size, kernel_args, sharedMemSize, 0);
        }
        
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipLaunchCooperativeKernel failed: %s\n", hipGetErrorString(cudaStatus));
            // Handle the error, for example, by cleaning up resources and exiting
            exit(1);
        }
        
        CHECK(hipDeviceSynchronize());
        timer.stop();
        totalMilliseconds += timer.elapsed();



        printf("Kernel invoked\n");

        // copying height, excess flow and residual flow values from device to host memory
        CHECK(hipMemcpy(cpu_height,gpu_height,V*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_excess_flow,gpu_excess_flow,V*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_fflows,gpu_fflows, E*sizeof(int),hipMemcpyDeviceToHost));

#ifdef WORKLOAD

        // Copy warp execution time from device to host
        copyFromStaticToArray<<<num_blocks, block_size>>>(gpu_warpExecutionTime, num_warps);
        hipDeviceSynchronize();

        CHECK(hipMemcpy(tempWarpExecution, gpu_warpExecutionTime, num_warps*sizeof(unsigned long long), hipMemcpyDeviceToHost));
        
        if (cur_iter < max_iter) {
            for (int i = 0; i < num_warps; i++) {
                cpuWarpExecution[i] += tempWarpExecution[i];
            }
        }
        cur_iter++;
#endif // WORKLOAD



        // printf("Before global relabel--------------------\n");


        //printExcessFlow(V,cpu_excess_flow);
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        //printf("Excess total : %d\n",*Excess_total);
        // perform the global_relabel routine on host
        // printf("Before global relabel, Excess total : %d\n",*Excess_total);
        
        // global_relabel(V, E, source,sink,cpu_height,cpu_excess_flow,
        //               cpu_offsets,cpu_destinations, cpu_capacities, cpu_fflows,
        //               Excess_total, mark, scanned);

        printf("Before global relabel--------------------\n");
        printf("Excess total: %d\n",*Excess_total);

        global_relabel_gpu(V, E, source, sink, 
                        cpu_height, cpu_excess_flow, cpu_offsets, cpu_destinations, cpu_capacities, cpu_fflows,
                        gpu_height, gpu_excess_flow, gpu_offsets, gpu_destinations, gpu_capacities, gpu_fflows,
                        Excess_total, mark, scanned);

        printf("After global relabel--------------------\n");
        // //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        printf("Excess total : %d\n",*Excess_total);
        
        // printf("Excess total : %d\n",*Excess_total);
        // printExcessFlow(V,cpu_excess_flow);

    }
    printf("Total kernel time: %.6f ms\n", totalMilliseconds);

#ifdef WORKLOAD
    printf("------------<< Workload Information >>------------\n");
    printf("#warps: %d\n", num_warps);
    printf("Warp execution time:\n");
    for (int i = 0; i < num_warps; i++) {
        printf("%llu ", cpuWarpExecution[i]);
    }
    printf("\n");

    // Free device buffer for warp execution time
    CHECK(hipFree(gpu_warpExecutionTime));

    // Free host buffer for warp execution time
    free(cpuWarpExecution);
    free(tempWarpExecution);

#endif // WORKLOAD

#ifdef TIME_BREAKDOWN
    // launch kernel to print device scanTime and backwardTime
    // printDeviceTime<<<num_blocks, block_size>>>();
    hipDeviceSynchronize();
    
    report_breakdown_data(totalMilliseconds);
    FinializeTimeBreakdown();
#endif /* TIME_BREAKDOWN */


}
