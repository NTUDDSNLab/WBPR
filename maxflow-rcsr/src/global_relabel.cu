#include"../include/parallel_graph.cuh"
#include "../include/utils.cuh"

void global_relabel(int V, int E, int source, int sink, int *cpu_height, int *cpu_excess_flow, 
                int *cpu_offsets, int *cpu_destinations, int* cpu_capacities, int* cpu_fflows, int* cpu_bflows, 
                int* cpu_roffsets, int* cpu_rdestinations, int* cpu_flow_idx,
                int *Excess_total, bool *mark, bool *scanned)
{
    for (int u = 0; u < V; u++) {
        // For all (u,v) belonging to E
        for (int i = cpu_offsets[u]; i < cpu_offsets[u + 1]; i++) {
            int v = cpu_destinations[i];

            if (cpu_height[u] > cpu_height[v] + 1) {
                
                // BUGS HERE! The cpu_excess_flow[u] might be smaller than cpu_fflow[i]
                // so we need to check if we can push more flow from u to v
                int flow;
                if (cpu_excess_flow[u] < cpu_fflows[i]) {
                    flow = cpu_excess_flow[u];
                } else {
                    flow = cpu_fflows[i];
                }

                cpu_excess_flow[u] -= flow;
                cpu_excess_flow[v] += flow;
                cpu_bflows[i] += flow;
                cpu_fflows[i] -= flow;
            }

        }
    }

    //printf("Prebfs\n");
    // performing backwards bfs from sink and assigning height values with each vertex's BFS tree level
    
    // declaring the Queue 
    std::list<int> Queue;

    // declaring variables to iterate over nodes for the backwards bfs and to store current tree level
    int x,y,current;
    
    // initialisation of the scanned array with false, before performing backwards bfs
    for(int i = 0; i < V; i++)
    {
        scanned[i] = false;
    }

    // Enqueueing the sink and set scan(sink) to true 
    Queue.push_back(sink);
    scanned[sink] = true;
    cpu_height[sink] = 0;

    // bfs routine and assigning of height values with tree level values
    while(!Queue.empty())
    {
        // dequeue
        x = Queue.front();
        Queue.pop_front();

        PRINTF("Global relabel: Dequeued: %d\n", x);

        // capture value of current level
        current = cpu_height[x];
        
        // increment current value
        current = current + 1;

        // For all (y,x) belonging to E_f (residual graph)
        // Scan reversed CSR but use the flow in the forward direction 
        for(int i = cpu_roffsets[x]; i < cpu_roffsets[x + 1]; i++)
        {
            y = cpu_rdestinations[i];
            int flow_index = cpu_flow_idx[i];
            PRINTF("Global relabel: (%d, %d)'s flow: %d\n",y, x, cpu_fflows[flow_index]);
            
            if (cpu_fflows[flow_index] > 0) {
                // if y is not scanned
                PRINTF("Global relabel: (%d, %d)'s flow > 0\n",x, y);
                if(scanned[y] == false)
                {
                    // assign current as height of y node
                    cpu_height[y] = current;

                    // mark scanned(y) as true
                    scanned[y] = true;

                    // Enqueue y
                    Queue.push_back(y);
                    PRINTF("Global relabel: Enqueued: %d\n", y);
                }
            }

        }

        for (int i = cpu_offsets[x]; i < cpu_offsets[x + 1]; i++) {
            y = cpu_destinations[i];
            int flow_index = i;
            PRINTF("Global relabel: (%d, %d)'s flow: %d\n",x, y, cpu_fflows[flow_index]);
            
            if (cpu_bflows[flow_index] > 0) {
                // if y is not scanned
                PRINTF("Global relabel: (%d, %d)'s flow > 0\n",x, y);
                if(scanned[y] == false)
                {
                    // assign current as height of y node
                    cpu_height[y] = current;

                    // mark scanned(y) as true
                    scanned[y] = true;

                    // Enqueue y
                    Queue.push_back(y);
                    PRINTF("Global relabel: Enqueued: %d\n", y);
                }
            }

        }

        // for(y = 0; y < V; y++)
        // {
        //     // for all (y,x) belonging to E_f (residual graph)
        //     if(cpu_rflowmtx[IDX(y,x)] > 0)
        //     {
        //         // if y is not scanned
        //         if(scanned[y] == false)
        //         {
        //             // assign current as height of y node
        //             cpu_height[y] = current;

        //             // mark scanned(y) as true
        //             scanned[y] = true;

        //             // Enqueue y
        //             Queue.push_back(y);
        //         }
        //     }
        // }

    }
    //printf("Pre check\n");
    // declaring and initialising boolean variable for checking if all nodes are relabeled
    bool if_all_are_relabeled = true;

    for(int i = 0; i < V; i++)
    {
        if(scanned[i] == false)
        {
            if_all_are_relabeled = false;
            break;
        }
    }

    // if not all nodes are relabeled
    if(if_all_are_relabeled == false)
    {
        // for all nodes
        for(int i = 0; i < V; i++)
        {
            // if i'th node is not marked or relabeled
            if( !( (scanned[i] == true) || (mark[i] == true) ) )
            {
                // mark i'th node
                mark[i] = true;

                /* decrement excess flow of i'th node from Excess_total
                    * This shows that i'th node is not scanned now and needs to be marked, thereby no more contributing to Excess_total
                    */
                PRINTF("Global relabel: %d is not scanned\n", i);
                *Excess_total = *Excess_total - cpu_excess_flow[i];
                // cpu_excess_flow[i] = 0;
                // cpu_height[i] = V;
                // printf("Removed excess flow from vertex: %d\n", i);
                //printf("Global relabel: Excess total: %d\n", *Excess_total);
            }
        }
    }
}


bool checkEnd(int V, int E, int source, int sink, int* cpu_excess_flow) {
    for (int u = 0; u < V; u++) {
        if (u != source && u != sink) {
            if (cpu_excess_flow[u] > 0) {
                return false;
            }
        }
    }
    return true;
}