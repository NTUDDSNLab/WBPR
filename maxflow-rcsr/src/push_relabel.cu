#include"../include/parallel_graph.cuh"
#include "../include/utils.cuh"

bool compare_excess_flow(int *new_excess_flow, int *old_excess_flow, int V)
{
    for(int i = 0; i < V; i++)
    {
        if (new_excess_flow[i] != old_excess_flow[i])
        {
            return false;
        }
    }
    return true;
}

void copy_excess_flow(int *new_excess_flow, int *old_excess_flow, int V)
{
    for(int i = 0; i < V; i++)
    {
        old_excess_flow[i] = new_excess_flow[i];
    }
}

void printExcessFlow(int V, int *excess_flow)
{
    printf("Excess flow values : \n");
    for(int i = 0; i < V; i++)
    {
        printf("%d ",excess_flow[i]);
    }
    printf("\n");
}


void push_relabel(int algo_type, int V, int E, int source, int sink, int *cpu_height, int *cpu_excess_flow, 
                int *cpu_offsets, int *cpu_destinations, int* cpu_capacities, int* cpu_fflows, int* cpu_bflows,
                int* cpu_roffsets, int* cpu_rdestinations, int* cpu_flow_idx,
                int *Excess_total, 
                int *gpu_height, int *gpu_excess_flow, 
                int *gpu_offsets, int* gpu_destinations, int* gpu_capacities, int* gpu_fflows, int* gpu_bflows,
                int* gpu_roffsets, int* gpu_rdestinations, int* gpu_flow_idx, 
                int* gpu_avq, int* gpu_cycle)
{
    /* Instead of checking for overflowing vertices(as in the sequential push relabel),
     * sum of excess flow values of sink and source are compared against Excess_total 
     * If the sum is lesser than Excess_total, 
     * it means that there is atleast one more vertex with excess flow > 0, apart from source and sink
     */

    /* declaring the mark and scan boolean arrays used in the global_relabel routine outside the while loop 
     * This is not to lose the mark values if it goes out of scope and gets redeclared in the next iteration 
     */
    
    bool *mark,*scanned;
    mark = (bool*)malloc(V*sizeof(bool));
    scanned = (bool*)malloc(V*sizeof(bool));




    CudaTimer timer;
    float totalMilliseconds = 0.0f;
    printf("Inside push_relabel\n");


    // Configure the GPU
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    dim3 num_blocks(deviceProp.multiProcessorCount * numBlocksPerSM);
    dim3 block_size(numThreadsPerBlock);
    // dim3 num_blocks(1);
    // dim3 block_size(64);

    // Calculate the usage of shared memory
    size_t sharedMemSize = 3 * block_size.x * sizeof(int);

#ifdef WORKLOAD
    // Caculate the total number of warps
    int num_warps = (block_size.x * num_blocks.x) / 32;
    
    // Allocate device buffer for warp execution time
    unsigned long long *gpu_warpExecutionTime;
    CHECK(hipMalloc((void**)&gpu_warpExecutionTime, num_warps*sizeof(unsigned long long)));

    // Allocate host buffer for warp execution time
    unsigned long long *cpuWarpExecution = (unsigned long long*)malloc(num_warps*sizeof(unsigned long long));
    unsigned long long *tempWarpExecution = (unsigned long long*)malloc(num_warps*sizeof(unsigned long long));
    for (int i = 0; i < num_warps; i++) {
        cpuWarpExecution[i] = 0;
    }
#endif // WORKLOAD

    // Print the configuration
    // Print GPU device name
    printf("GPU Device: %s\n", deviceProp.name);
    printf("Number of blocks: %d\n", num_blocks.x);
    printf("Number of threads per block: %d\n", block_size.x);
    printf("Shared memory size: %lu\n", sharedMemSize);

    void* original_kernel_args[] = {&V, &source, &sink, &gpu_height, &gpu_excess_flow, 
                        &gpu_offsets, &gpu_destinations, &gpu_capacities, &gpu_fflows, &gpu_bflows, 
                        &gpu_roffsets, &gpu_rdestinations, &gpu_flow_idx};


    void* kernel_args[] = {&V, &source, &sink, &gpu_height, &gpu_excess_flow, 
                        &gpu_offsets, &gpu_destinations, &gpu_capacities, &gpu_fflows, &gpu_bflows, 
                        &gpu_roffsets, &gpu_rdestinations, &gpu_flow_idx, 
                        &gpu_avq, &gpu_cycle};


    // initialising mark values to false for all nodes
    for(int i = 0; i < V; i++)
    {
        mark[i] = false;
    }
    //for (int i = 0; i < 3; i++)
    while((cpu_excess_flow[source] + cpu_excess_flow[sink]) < *Excess_total)
    {
        printf("cpu_excess_flow[source]: %d, cpu_excess_flow[sink]: %d\n",cpu_excess_flow[source], cpu_excess_flow[sink]);

        //printf("gpu_excess_flow[source]: %d, gpu_excess_flow[sink]: %d\n",gpu_excess_flow[source], gpu_excess_flow[sink]);
        // copying height values to CUDA device global memory
        CHECK(hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpu_excess_flow, cpu_excess_flow, V*sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpu_fflows, cpu_fflows, E*sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpu_bflows, cpu_bflows, E*sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemset(gpu_cycle, V, sizeof(int))); // Reset the gpu_cycle to V


        printf("Invoking kernel\n");

        hipError_t cudaStatus;
        timer.start();
        if (algo_type == 0) {
            // Thread-centric approach
            cudaStatus = hipLaunchCooperativeKernel((void*)push_relabel_kernel, num_blocks, block_size, original_kernel_args, sharedMemSize, 0);
        } else {
            // Vertex-centric approach
            cudaStatus = hipLaunchCooperativeKernel((void*)coop_push_relabel_kernel, num_blocks, block_size, kernel_args, sharedMemSize, 0);
        }
        
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipLaunchCooperativeKernel failed: %s\n", hipGetErrorString(cudaStatus));
            // Handle the error, for example, by cleaning up resources and exiting
            exit(1);
        }
        
        hipDeviceSynchronize();
        timer.stop();
        totalMilliseconds += timer.elapsed();



        printf("Kernel invoked\n");

        // copying height, excess flow and residual flow values from device to host memory
        CHECK(hipMemcpy(cpu_height,gpu_height,V*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_excess_flow,gpu_excess_flow,V*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_fflows,gpu_fflows, E*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_bflows,gpu_bflows, E*sizeof(int),hipMemcpyDeviceToHost));

#ifdef WORKLOAD

        // Copy warp execution time from device to host
        copyFromStaticToArray<<<num_blocks, block_size>>>(gpu_warpExecutionTime, num_warps);
        hipDeviceSynchronize();

        CHECK(hipMemcpy(tempWarpExecution, gpu_warpExecutionTime, num_warps*sizeof(unsigned long long), hipMemcpyDeviceToHost));
        for (int i = 0; i < num_warps; i++) {
            cpuWarpExecution[i] += tempWarpExecution[i];
        }
#endif // WORKLOAD



        // printf("Before global relabel--------------------\n");


        //printExcessFlow(V,cpu_excess_flow);
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        //printf("Excess total : %d\n",*Excess_total);
        // perform the global_relabel routine on host
        // printf("Before global relabel, Excess total : %d\n",*Excess_total);

        global_relabel(V, E, source,sink,cpu_height,cpu_excess_flow,
                      cpu_offsets,cpu_destinations, cpu_capacities, cpu_fflows, cpu_bflows,
                      cpu_roffsets, cpu_rdestinations, cpu_flow_idx,
                      Excess_total, mark, scanned);

        printf("After global relabel--------------------\n");
        // //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        printf("Excess total : %d\n",*Excess_total);
        
        // printf("Excess total : %d\n",*Excess_total);
        // printExcessFlow(V,cpu_excess_flow);

    }
    printf("Total kernel time: %.6f ms\n", totalMilliseconds);

#ifdef WORKLOAD
    printf("------------<< Workload Information >>------------\n");
    printf("#warps: %d\n", num_warps);
    printf("Warp execution time:\n");
    for (int i = 0; i < num_warps; i++) {
        printf("%llu ", cpuWarpExecution[i]);
    }
    printf("\n");

    // Free device buffer for warp execution time
    CHECK(hipFree(gpu_warpExecutionTime));

    // Free host buffer for warp execution time
    free(cpuWarpExecution);
    free(tempWarpExecution);

#endif // WORKLOAD


}
